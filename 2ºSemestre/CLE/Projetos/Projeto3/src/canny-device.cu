#include "hip/hip_runtime.h"

// CLE 24'25

typedef int pixel_t;

/**
 * @brief CUDA Kernel to perform 2D convolution on an image.
 * @param in Pointer to the input image data.
 * @param out Pointer to the output image data.
 * @param kernel Pointer to the convolution kernel.
 * @param nx Width of the image.
 * @param ny Height of the image.
 * @param kn Size of the kernel
 */
__global__ void convolutionKernel(
    const pixel_t *in, pixel_t *out, const float *kernel,
    const int nx, const int ny, const int kn)
{
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;

    const int khalf = kn / 2;

    // Ensure the thread is within bounds of the image
    if (m >= khalf && m < nx - khalf && n >= khalf && n < ny - khalf)
    {
        float pixel = 0.0f;

        for (int j = -khalf; j <= khalf; j++)
        {
            for (int i = -khalf; i <= khalf; i++)
            {
                int idx = (n - j) * nx + (m - i);         // Index in the input image
                int kid = (j + khalf) * kn + (i + khalf); // Index in the kernel
                pixel += in[idx] * kernel[kid];
            }
        }

        out[n * nx + m] = (pixel_t)pixel;
    }
}

/**
 * @brief CUDA Kernel to perform non-maximum suppression on the gradient magnitude image.
 * @param after_Gx Pointer to the image gradient along x.
 * @param after_Gy Pointer to the image gradient along y.
 * @param G Pointer to the gradient magnitude image.
 * @param nms Pointer to the output image after non-maximum suppression.
 * @param nx Width of the image.
 * @param ny Height of the image.
 */
__global__ void nonMaximumSuppressionKernel(
    const pixel_t *after_Gx, pixel_t *after_Gy,
    const pixel_t *G, pixel_t *nms,
    const int nx, const int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Skip border pixels
    if (i >= 1 && i < nx - 1 && j >= 1 && j < ny - 1)
    {
        int c = i + nx * j;
        const int nn = c - nx;  // North
        const int ss = c + nx;  // South
        const int ww = c + 1;   // West
        const int ee = c - 1;   // East
        const int nw = nn + 1;  // North-West
        const int ne = nn - 1;  // North-East
        const int sw = ss + 1;  // South-West
        const int se = ss - 1;  // South-East

        const float dir = (float)(fmodf(atan2f((float)after_Gy[c], (float)after_Gx[c]) + M_PI, M_PI) / M_PI) * 8;

        if (((dir <= 1 || dir > 7) && G[c] > G[ee] && G[c] > G[ww]) || // 0 deg (horizontal)
            ((dir > 1 && dir <= 3) && G[c] > G[nw] && G[c] > G[se]) || // 45 deg (diagonal)
            ((dir > 3 && dir <= 5) && G[c] > G[nn] && G[c] > G[ss]) || // 90 deg (vertical)
            ((dir > 5 && dir <= 7) && G[c] > G[ne] && G[c] > G[sw]))   // 135 deg (diagonal)
        {
            nms[c] = G[c];  // Keep local maximum
        }
        else
        {
            nms[c] = 0; // Suppress non-maximum
        }
    }
}

/**
 * @brief CUDA Kernel to mark the first edges in the non-maximum suppressed image.
 * @param nms Pointer to the non-maximum suppressed image.
 * @param reference Pointer to the output image marking first edges.
 * @param nx Width of the image.
 * @param ny Height of the image.
 * @param tmax High threshold for marking edges.
 */
__global__ void firstEdgesKernel(
    const pixel_t *nms, pixel_t *reference,
    const int nx, const int ny, const int tmax)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Skip border pixels
    if (i >= 1 && i < nx - 1 && j >= 1 && j < ny - 1)
    {
        int c = i + nx * j;
        if (nms[c] >= tmax)
        {
            reference[c] = 255; // Strong edge
        }
        else
        {
            reference[c] = 0; // Weak edge
        }
    }
}

/**
 * @brief CUDA kernel to perform edge tracking by hysteresis in Canny edge detection.
 * @param nms Pointer to the non-maximum suppressed image.
 * @param reference Pointer to the output image after hysteresis thresholding.
 * @param nx Width of the image.
 * @param ny Height of the image.
 * @param tmin Low threshold for weak edge detection
 * @param changed Pointer to a boolean flag indicating if any changes were made.
 */
__global__ void hysteresisKernel(
    const pixel_t *nms, pixel_t *reference,
    const int nx, const int ny,
    const int tmin, bool *changed)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Skip border pixels
    if (i >= 1 && i < nx - 1 && j >= 1 && j < ny - 1)
    {
        int t = i + j * nx;

        int nbs[8];
        nbs[0] = t - nx;     // North
        nbs[1] = t + nx;     // South
        nbs[2] = t + 1;      // West
        nbs[3] = t - 1;      // East
        nbs[4] = nbs[0] + 1; // North-West
        nbs[5] = nbs[0] - 1; // North-East
        nbs[6] = nbs[1] + 1; // South-West
        nbs[7] = nbs[1] - 1; // South-East

        if (nms[t] >= tmin && reference[t] == 0)
        {
            for (int k = 0; k < 8; k++)
            {
                if (reference[nbs[k]] >= tmin)
                {
                    reference[t] = 255; // Promote to strong edge
                    *changed = true;
                }
            }
        }
    }
}

/**
 * @brief CUDA kernel to generate a Gaussian kernel for image smoothing.
 * @param kernel Pointer to the output kernel array.
 * @param n Size of the kernel (n x n).
 * @param sigma Standard deviation for the Gaussian distribution.
 */
__global__ void gaussianKernel(
    float *kernel, const int n, const float sigma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n)
    {
        const float mean = (float)floor(n / 2.0);
        int c = i + n * j;
        kernel[c] = expf(-0.5f * (powf((i - mean) / sigma, 2.0f) + powf((j - mean) / sigma, 2.0f))) /
                    (2 * M_PI * sigma * sigma);
    }
}

/**
 * @brief CUDA kernel to find the minimum and maximum pixel values in an image using shared memory reduction.
 * @param data Pointer to the input image data.
 * @param min_vals Pointer to the output array for minimum values.
 * @param max_vals Pointer to the output array for maximum values.
 * @param nx Width of the image.
 * @param ny Height of the image.
 */
__global__ void minMaxKernel(
    const pixel_t *data, pixel_t *min_vals, pixel_t *max_vals,
    const int nx, const int ny)
{
    extern __shared__ pixel_t sdata[];
    pixel_t *s_min = sdata;
    pixel_t *s_max = &sdata[blockDim.x];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Initialize shared memory
    if (i < nx * ny)
    {
        s_min[tid] = data[i];
        s_max[tid] = data[i];
    }
    else
    {
        s_min[tid] = INT_MAX;
        s_max[tid] = INT_MIN;
    }

    __syncthreads();

    // Reduce to find min and max
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s && i + s < nx * ny)
        {
            if (s_min[tid + s] < s_min[tid])
                s_min[tid] = s_min[tid + s];
            if (s_max[tid + s] > s_max[tid])
                s_max[tid] = s_max[tid + s];
        }
        __syncthreads();
    }

    // First thread writes results to global memory
    if (tid == 0)
    {
        atomicMin(&min_vals[blockIdx.x], s_min[0]);
        atomicMax(&max_vals[blockIdx.x], s_max[0]);
    }
}

/**
 * @brief CUDA kernel to find the minimum and maximum pixel values in an image without using shared memory.
 * @param data Pointer to the input image data.
 * @param global_min Pointer to the global minimum value.
 * @param global_max Pointer to the global maximum value.
 * @param nx Width of the image.
 * @param ny Height of the image.
 */
__global__ void minMaxKernel_no_shared(
    const pixel_t *data, pixel_t *global_min, pixel_t *global_max,
    const int nx, const int ny
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int size = nx * ny;

    if (i < size)
    {
        pixel_t val = data[i];

        atomicMin(global_min, val);
        atomicMax(global_max, val);
    }
}

/**
 * @brief CUDA function to normalize pixel values in an image based on the [0, 255] range.
 * @param data Pointer to the input image data to be normalized.
 * @param nx Width of the image.
 * @param ny Height of the image.
 * @param kn Size of the kernel used for normalization.
 * @param min_val Minimum pixel value in the image.
 * @param max_val Maximum pixel value in the image.
 */
__global__ void normalizeKernel(
    pixel_t *data, const int nx, const int ny,
    const int kn, const pixel_t min_val, const pixel_t max_val)
{
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;

    const int khalf = kn / 2;

    // Only normalize pixels not on the border
    if (m >= khalf && m < nx - khalf && n >= khalf && n < ny - khalf)
    {
        int idx = m + nx * n;
        pixel_t pixel = 255 * ((int)data[idx] - (float)min_val) / ((float)max_val - (float)min_val);
        data[idx] = pixel;
    }
}

/**
 * @brief CUDA kernel to merge gradients from Sobel operators into a single gradient magnitude image.
 * @param G Pointer to the output gradient magnitude image.
 * @param Gx Pointer to the gradient along x.
 * @param Gy Pointer to the gradient along y.
 * @param nx Width of the image.
 * @param ny Height of the image.
 */
__global__ void mergeGradientsKernel(
    pixel_t *G, const pixel_t *Gx, const pixel_t *Gy,
    const int nx, const int ny)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < nx * ny)
    {
        int i = idx % nx;
        int j = idx / nx;

        // Only compute for non-border pixels
        if (i >= 1 && i < nx - 1 && j >= 1 && j < ny - 1)
        {
            G[idx] = (pixel_t)(hypotf((float)Gx[idx], (float)Gy[idx]));
        }
    }
}

/**
 * @brief CUDA function to find the minimum and maximum pixel values in an image using shared memory reduction.
 * @param d_data Pointer to the input image data.
 * @param nx Width of the image.
 * @param ny Height of the image.
 * @param min_val Pointer to host memory to store the minimum value found.
 * @param max_val Pointer to host memory to store the maximum value found.
 */
void cudaMinMax(
    const pixel_t *d_data, const int nx, const int ny,
    pixel_t *min_val, pixel_t *max_val)
{
    printf("Using shared memory approach for min/max calculation.\n");
    int num_elements = nx * ny;
    int block_size = 256;
    int num_blocks = (num_elements + block_size - 1) / block_size;

    pixel_t *d_min_vals, *d_max_vals;
    pixel_t *h_min_vals, *h_max_vals;

    cudaSafeCall(hipMalloc(&d_min_vals, num_blocks * sizeof(pixel_t)));
    cudaSafeCall(hipMalloc(&d_max_vals, num_blocks * sizeof(pixel_t)));

    h_min_vals = (pixel_t *)malloc(num_blocks * sizeof(pixel_t));
    h_max_vals = (pixel_t *)malloc(num_blocks * sizeof(pixel_t));

    minMaxKernel<<<num_blocks, block_size, 2 * block_size * sizeof(pixel_t)>>>(
        d_data, d_min_vals, d_max_vals, nx, ny);

    cudaSafeCall(hipMemcpy(h_min_vals, d_min_vals, num_blocks * sizeof(pixel_t), hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(h_max_vals, d_max_vals, num_blocks * sizeof(pixel_t), hipMemcpyDeviceToHost));

    *min_val = h_min_vals[0];
    *max_val = h_max_vals[0];

    for (int i = 1; i < num_blocks; i++)
    {
        if (h_min_vals[i] < *min_val)
            *min_val = h_min_vals[i];
        if (h_max_vals[i] > *max_val)
            *max_val = h_max_vals[i];
    }

    hipFree(d_min_vals);
    hipFree(d_max_vals);
    free(h_min_vals);
    free(h_max_vals);
}

/**
 * @brief CUDA function to find the minimum and maximum pixel values in an image without using shared memory.
 * @param d_data Pointer to the input image data.
 * @param nx Width of the image.
 * @param ny Height of the image.
 * @param min_val Pointer to host memory to store the minimum value found.
 * @param max_val Pointer to host memory to store the maximum value found.
 */
void cudaMinMax_noShared(
    const pixel_t *d_data, const int nx, const int ny,
    pixel_t *min_val, pixel_t *max_val)
{
    printf("Using no shared memory approach for min/max calculation.\n");
    int num_elements = nx * ny;
    int block_size = 256;
    int num_blocks = (num_elements + block_size - 1) / block_size;

    // Allocate device-side memory for min and max values
    pixel_t *d_min, *d_max;
    cudaSafeCall(hipMalloc(&d_min, sizeof(pixel_t)));
    cudaSafeCall(hipMalloc(&d_max, sizeof(pixel_t)));

    pixel_t init_min = 0x7FFFFFFF; // INT_MAX
    pixel_t init_max = 0x80000000; // INT_MIN
    cudaSafeCall(hipMemcpy(d_min, &init_min, sizeof(pixel_t), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(d_max, &init_max, sizeof(pixel_t), hipMemcpyHostToDevice));

    minMaxKernel_no_shared<<<num_blocks, block_size>>>(
        d_data, d_min, d_max, nx, ny);

    cudaSafeCall(hipMemcpy(min_val, d_min, sizeof(pixel_t), hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(max_val, d_max, sizeof(pixel_t), hipMemcpyDeviceToHost));

    hipFree(d_min);
    hipFree(d_max);
}

// canny edge detector code to run on the GPU
void cannyDevice(const int *h_idata, const int w, const int h,
                 const int tmin, const int tmax,
                 const float sigma, int *h_odata, 
                 bool use_shared_memory)
{
    const int nx = w;
    const int ny = h;
    const int size = nx * ny * sizeof(pixel_t);

    // Device memory allocation
    pixel_t *d_input, *d_temp, *d_after_Gx, *d_after_Gy, *d_G, *d_nms, *d_reference;
    float *d_kernel, *d_Gx, *d_Gy;

    cudaSafeCall(hipMalloc(&d_input, size));
    cudaSafeCall(hipMalloc(&d_temp, size));
    cudaSafeCall(hipMalloc(&d_after_Gx, size));
    cudaSafeCall(hipMalloc(&d_after_Gy, size));
    cudaSafeCall(hipMalloc(&d_G, size));
    cudaSafeCall(hipMalloc(&d_nms, size));
    cudaSafeCall(hipMalloc(&d_reference, size));

    // Copy input data to device
    cudaSafeCall(hipMemcpy(d_input, h_idata, size, hipMemcpyHostToDevice));
    cudaSafeCall(hipMemset(d_reference, 0, size));

    // Define thread block and grid dimensions
    dim3 block(8, 8);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // Apply Gaussian filter
    const int n = 2 * (int)(2 * sigma) + 3;
    cudaSafeCall(hipMalloc(&d_kernel, n * n * sizeof(float)));

    dim3 block_kernel(16, 16);
    dim3 grid_kernel((n + block_kernel.x - 1) / block_kernel.x, (n + block_kernel.y - 1) / block_kernel.y);
    gaussianKernel<<<grid_kernel, block_kernel>>>(d_kernel, n, sigma);
    cudaCheckMsg("Gaussian kernel generation failed");

    convolutionKernel<<<grid, block>>>(d_input, d_temp, d_kernel, nx, ny, n);
    cudaCheckMsg("Gaussian convolution failed");

    // Normalize gaussian result
    pixel_t min_val, max_val;

    if (use_shared_memory) cudaMinMax(d_temp, nx, ny, &min_val, &max_val);
    else cudaMinMax_noShared(d_temp, nx, ny, &min_val, &max_val);

    normalizeKernel<<<grid, block>>>(d_temp, nx, ny, n, min_val, max_val);
    cudaCheckMsg("Gaussian normalization failed");


    // Sobel operators
    float h_Gx[9] = {
        -1, 0, 1,
        -2, 0, 2,
        -1, 0, 1};

    float h_Gy[9] = {
        1, 2, 1,
        0, 0, 0,
        -1, -2, -1};

    cudaSafeCall(hipMalloc(&d_Gx, 9 * sizeof(float)));
    cudaSafeCall(hipMalloc(&d_Gy, 9 * sizeof(float)));
    cudaSafeCall(hipMemcpy(d_Gx, h_Gx, 9 * sizeof(float), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(d_Gy, h_Gy, 9 * sizeof(float), hipMemcpyHostToDevice));

    // Gradient along x
    convolutionKernel<<<grid, block>>>(d_temp, d_after_Gx, d_Gx, nx, ny, 3);
    cudaCheckMsg("Gradient X convolution failed");

    // Gradient along y
    convolutionKernel<<<grid, block>>>(d_temp, d_after_Gy, d_Gy, nx, ny, 3);
    cudaCheckMsg("Gradient Y convolution failed");

    // Merge gradients
    int num_threads = 256;
    int num_blocks = (nx * ny + num_threads - 1) / num_threads;

    mergeGradientsKernel<<<num_blocks, num_threads>>>(d_G, d_after_Gx, d_after_Gy, nx, ny);
    cudaCheckMsg("Gradient merge failed");

    // Non-maximum suppression
    nonMaximumSuppressionKernel<<<grid, block>>>(d_after_Gx, d_after_Gy, d_G, d_nms, nx, ny);
    cudaCheckMsg("Non-maximum suppression failed");

    // First edges
    firstEdgesKernel<<<grid, block>>>(d_nms, d_reference, nx, ny, tmax);
    cudaCheckMsg("First edges failed");

    // Hysteresis edges
    bool *d_changed;
    bool h_changed;
    cudaSafeCall(hipMalloc(&d_changed, sizeof(bool)));

    do
    {
        h_changed = false;
        cudaSafeCall(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));

        hysteresisKernel<<<grid, block>>>(d_nms, d_reference, nx, ny, tmin, d_changed);
        cudaCheckMsg("Hysteresis failed");

        cudaSafeCall(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
    } while (h_changed);

    // Copy result to host
    cudaSafeCall(hipMemcpy(h_odata, d_reference, size, hipMemcpyDeviceToHost));

    // Clean up
    hipFree(d_input);
    hipFree(d_temp);
    hipFree(d_after_Gx);
    hipFree(d_after_Gy);
    hipFree(d_G);
    hipFree(d_nms);
    hipFree(d_reference);
    hipFree(d_kernel);
    hipFree(d_Gx);
    hipFree(d_Gy);
    hipFree(d_changed);
}